#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/center_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Compute_distance_data_gpu(int nthreads, const int K, const Dtype* bottom,
	      const Dtype* label, const Dtype* center, Dtype* distance) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int m = index / K;
    int k = index % K;
    const int label_value = static_cast<int>(label[m]);
    // distance(i) = x(i) - c_{y(i)}
    distance[index] = bottom[index] - center[label_value * K + k];
  }
}

template <typename Dtype>
__global__ void Compute_center_diff_gpu(int nthreads, const int M, const int K, 
        const Dtype* label, const Dtype* distance, Dtype* variation_sum, 
        Dtype* center_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int count = 0;
    for (int m = 0; m < M; m++) {
      const int label_value = static_cast<int>(label[m]);
      if (label_value == index) {
        count++;
        for (int k = 0; k < K; k++) {
          variation_sum[index * K + k] -= distance[m * K + k];
        }
      }
    }
    for (int k = 0; k < K; k++) {
      center_diff[index * K + k] = variation_sum[index * K + k] /(count + (Dtype)1.);
    }
  }
}


template <typename Ftype, typename Btype>
void CenterLossLayer<Ftype, Btype>::Forward_gpu(
    const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  int nthreads = M_ * K_;
  Compute_distance_data_gpu<Ftype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, 
                                K_,
                                bottom[0]->gpu_data<Ftype>(),
                                bottom[1]->gpu_data<Ftype>(),
                                this->blobs_[0]->template gpu_data<Ftype>(),
                                distance_.template mutable_gpu_data<Ftype>());
  Ftype dot;
  caffe_gpu_dot<Ftype>(M_ * K_, 
    distance_.template gpu_data<Ftype>(),
    distance_.template gpu_data<Ftype>(),
    &dot);
  Ftype loss = dot / M_ / Ftype(2);
  top[0]->mutable_cpu_data<Ftype>()[0] = loss;
}

template <typename Ftype, typename Btype>
void CenterLossLayer<Ftype, Btype>::Backward_gpu(
    const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  int nthreads = N_;
  caffe_gpu_set<Btype>(N_ * K_, (Btype)0., variation_sum_.template  mutable_cpu_data<Btype>());
  Compute_center_diff_gpu<Btype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(
      nthreads,
      M_, 
      K_,
      bottom[1]->template gpu_data<Btype>(),
      distance_.template gpu_data<Btype>(), 
      variation_sum_.template mutable_cpu_data<Btype>(),
      this->blobs_[0]->template mutable_gpu_diff<Btype>());

  if (propagate_down[0]) {
    caffe_gpu_scale<Btype>(M_ * K_,
		top[0]->template cpu_diff<Btype>()[0] / M_, 
        distance_.template gpu_data<Btype>(),
        bottom[0]->template mutable_gpu_diff<Btype>());
  }
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(CenterLossLayer);

}  // namespace caffe
