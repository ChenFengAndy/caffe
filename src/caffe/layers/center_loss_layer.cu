#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/center_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Compute_distance_data_gpu(int nthreads, const int K, const Dtype* bottom,
	      const Dtype* label, const Dtype* center, Dtype* distance) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int m = index / K;
    int k = index % K;
    const int label_value = static_cast<int>(label[m]);
    // distance(i) = x(i) - c_{y(i)}
    distance[index] = bottom[index] - center[label_value * K + k];
  }
}

//Andy work on each input element
template <typename Dtype>
__global__ void Compute_center_diff_gpu_per_sample(int nthreads, const int M, const int K, 
        const Dtype* label, const float* distance, float* variation_sum) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    //for (int m = 0; m < M; m++) {
      const int label_value = static_cast<int>(label[index]);
      //if (label_value == index) {
        //count++;
	    //atomicAdd(&count[label_value], 1);
        for (int k = 0; k < K; k++) {
          atomicAdd(&variation_sum[label_value * K + k], -distance[index * K + k]);
        }
      //}
    //}
	
  }
}



//Andy work on each class 
//need a count table to store the samples per class
template <typename Dtype>
__global__ void Compute_center_diff_gpu_per_class(int nthreads, const int M, const int K, 
        const int* count_table, const Dtype* variation_sum, 
        Dtype* center_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
  	const int count = static_cast<int>(count_table[index]);
    for (int k = 0; k < K; k++) {
      center_diff[index * K + k] = variation_sum[index * K + k] /(count + (Dtype)1.);
    }
  }
}

//create count table
template <typename Dtype>
__global__ void create_count_table_gpu(int nthreads, const int M, 
        const Dtype* label, int* count_table) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int count = 0;
    for (int m = 0; m < M; m++) {
      const int label_value = static_cast<int>(label[m]);
      if (label_value == index) {
        count++;
      }
    }
	count_table[index] = count;
  }
}


template <typename Dtype>
__global__ void Compute_center_diff_gpu(int nthreads, const int M, const int K, 
        const Dtype* label, const Dtype* distance, Dtype* variation_sum, 
        Dtype* center_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int count = 0;
    for (int m = 0; m < M; m++) {
      const int label_value = static_cast<int>(label[m]);
      if (label_value == index) {
        count++;
        for (int k = 0; k < K; k++) {
          variation_sum[index * K + k] -= distance[m * K + k];
        }
      }
    }
    for (int k = 0; k < K; k++) {
      center_diff[index * K + k] = variation_sum[index * K + k] /(count + (Dtype)1.);
    }
  }
}


template <typename Ftype, typename Btype>
void CenterLossLayer<Ftype, Btype>::Forward_gpu(
    const vector<Blob*>& bottom,
    const vector<Blob*>& top) {
  int nthreads = M_ * K_;
  Compute_distance_data_gpu<Ftype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, 
                                K_,
                                bottom[0]->gpu_data<Ftype>(),
                                bottom[1]->gpu_data<Ftype>(),
                                this->blobs_[0]->template gpu_data<Ftype>(),
                                distance_.template mutable_gpu_data<Ftype>());
  Ftype dot;
  caffe_gpu_dot<Ftype>(M_ * K_, 
    distance_.template gpu_data<Ftype>(),
    distance_.template gpu_data<Ftype>(),
    &dot);
  Ftype loss = dot / M_ / Ftype(2);
  top[0]->mutable_cpu_data<Ftype>()[0] = loss;
  
}

template <typename Ftype, typename Btype>
void CenterLossLayer<Ftype, Btype>::Backward_gpu(
    const vector<Blob*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob*>& bottom) {
  int nthreads = N_;

  if(!COUNTED_) {
  	//count_ table not ready
  	
	 create_count_table_gpu<Btype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(
      nthreads,
      M_, 
      bottom[1]->template gpu_data<Btype>(),
      count_.template mutable_cpu_data<int>() 
      );
	 COUNTED_ = true;
  }
  caffe_gpu_set<Btype>(N_ * K_, (Btype)0., variation_sum_.template  mutable_cpu_data<Btype>());
  /*
  Compute_center_diff_gpu<Btype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(
      nthreads,
      M_, 
      K_,
      bottom[1]->template gpu_data<Btype>(),
      distance_.template gpu_data<Btype>(), 
      variation_sum_.template mutable_cpu_data<Btype>(),
      this->blobs_[0]->template mutable_gpu_diff<Btype>());
  */
  nthreads = M_;  
  Compute_center_diff_gpu_per_sample<Btype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(
      nthreads,
      M_, 
      K_,
      bottom[1]->template gpu_data<Btype>(),
      distance_.template gpu_data<float>(), 
      variation_sum_.template mutable_cpu_data<float>()
      );
  nthreads = N_;  
  Compute_center_diff_gpu_per_class<Btype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(
      nthreads,
      M_, 
      K_,
      count_.template gpu_data<int>(),
      variation_sum_.template gpu_data<Btype>(),
      this->blobs_[0]->template mutable_gpu_diff<Btype>());


  
  if (propagate_down[0]) {
    caffe_gpu_scale<Btype>(M_ * K_,
		top[0]->template cpu_diff<Btype>()[0] / M_, 
        distance_.template gpu_data<Btype>(),
        bottom[0]->template mutable_gpu_diff<Btype>());
  }
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
}

INSTANTIATE_LAYER_GPU_FUNCS_FB(CenterLossLayer);

}  // namespace caffe
